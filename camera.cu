#include "hip/hip_runtime.h"
//
// Copyright (c) 2021, NVIDIA CORPORATION. All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions
// are met:
//  * Redistributions of source code must retain the above copyright
//    notice, this list of conditions and the following disclaimer.
//  * Redistributions in binary form must reproduce the above copyright
//    notice, this list of conditions and the following disclaimer in the
//    documentation and/or other materials provided with the distribution.
//  * Neither the name of NVIDIA CORPORATION nor the names of its
//    contributors may be used to endorse or promote products derived
//    from this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
// EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
// PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
// CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
// EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
// PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
// PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
// OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
// (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
//

#include <hip/hip_vector_types.h>
#include <optix_device.h>
#include "optixWhitted.h"
#include "random.h"
#include "helpers.h"
#include <cuda/helpers.h>


extern "C" {
__constant__ Params params;
}

extern "C" __global__ void __raygen__pinhole_camera()
{
    const uint3 idx = optixGetLaunchIndex();

    const CameraData* camera = (CameraData*) optixGetSbtDataPointer();

    const unsigned int image_index = params.width * idx.y + idx.x;

    float2 subpixel_jitter = make_float2(0.5f, 0.5f) ;

    float2 d = ((make_float2(idx.x, idx.y) + subpixel_jitter) / make_float2(params.width, params.height)) * 2.f - 1.f;
    float3 ray_origin = camera->eye;
    float3 ray_direction = normalize(d.x*camera->U + d.y*camera->V + camera->W);

    float3 payload_rgb = make_float3(1e17f, 0.f, 0.f);

    optixTrace(
        params.handle,                     // handle
        ray_origin,                         // float3 rayOrigin
        ray_direction,                      // float3 rayDirection
        0.f,                                 // float tmin
        1e16f,                              // float tmax
        0.0f,                               // float rayTime
        OptixVisibilityMask( 1 ),
        OPTIX_RAY_FLAG_CULL_BACK_FACING_TRIANGLES,
        1,                          // SBT offset (1 = CH2)
        RAY_TYPE_COUNT,             // SBT stride
        RAY_TYPE_RADIANCE,          // missSBTIndex 
        float3_as_args(payload_rgb));

    float distanceMin = payload_rgb.x;
    payload_rgb.x = 0.f;
    


    optixTrace(
        params.handle,                     // handle
        ray_origin,                         // float3 rayOrigin
        ray_direction,                      // float3 rayDirection
        0.f,                                 // float tmin
        1e16f,                              // float tmax
        0.0f,                               // float rayTime
        OptixVisibilityMask( 1 ),
        OPTIX_RAY_FLAG_CULL_FRONT_FACING_TRIANGLES,
        1,                          // SBT offset (1 = CH2)
        RAY_TYPE_COUNT,             // SBT stride
        RAY_TYPE_RADIANCE,          // missSBTIndex 
        float3_as_args(payload_rgb));

    float distanceMax = payload_rgb.x;
    payload_rgb.x = 0.f;

    float3 position;

    for (float t = distanceMin; t < distanceMax; t +=0.05f)
    {
        position = ray_origin + t * ray_direction;

        optixTrace(
        params.handle,                     // handle
        position,                         // float3 rayOrigin
        ray_direction,                      // float3 rayDirection
        0.f,                                 // float tmin
        1e16f,                              // float tmax
        0.0f,                               // float rayTime
        OptixVisibilityMask( 1 ),
        OPTIX_RAY_FLAG_NONE,
        0,                          // SBT offset (1 = CH2)
        RAY_TYPE_COUNT,             // SBT stride
        RAY_TYPE_RADIANCE,          // missSBTIndex 
        float3_as_args(payload_rgb));

    }

    
    params.frame_buffer[image_index] = make_color( payload_rgb );
  

}
