#include "hip/hip_runtime.h"
//
// Copyright (c) 2021, NVIDIA CORPORATION. All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions
// are met:
//  * Redistributions of source code must retain the above copyright
//    notice, this list of conditions and the following disclaimer.
//  * Redistributions in binary form must reproduce the above copyright
//    notice, this list of conditions and the following disclaimer in the
//    documentation and/or other materials provided with the distribution.
//  * Neither the name of NVIDIA CORPORATION nor the names of its
//    contributors may be used to endorse or promote products derived
//    from this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
// EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
// PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
// CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
// EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
// PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
// PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
// OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
// (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
//

#include <hip/hip_vector_types.h>
#include <optix_device.h>
#include "optixWhitted.h"
#include "random.h"
#include "helpers.h"
#include <cuda/helpers.h>

extern "C" {
__constant__ Params params;
}

extern "C" __global__ void __raygen__pinhole_camera()
{
    const uint3 idx = optixGetLaunchIndex();
    const uint3 dim = optixGetLaunchDimensions();

    const CameraData* camera = (CameraData*) optixGetSbtDataPointer();

    const unsigned int image_index = params.width * idx.y + idx.x;
    unsigned int       seed        = tea<16>( image_index, params.subframe_index );

    // Subpixel jitter: send the ray through a different position inside the pixel each time,
    // to provide antialiasing. The center of each pixel is at fraction (0.5,0.5)
    float2 subpixel_jitter = params.subframe_index == 0 ?
        make_float2(0.5f, 0.5f) : make_float2(rnd( seed ), rnd( seed ));

    float2 d = ((make_float2(idx.x, idx.y) ) / make_float2(params.width, params.height)) * 2.f - 1.f;
    float3 ray_origin = camera->eye;
    float3 ray_direction = normalize(d.x*camera->U + d.y*camera->V + camera->W);

    float3 payload_rgb = make_float3( 0.f, 0.f, 0.f);

    //OPTIX_RAY_FLAG_CULL_FRONT_FACING_TRIANGLES,
    //OPTIX_RAY_FLAG_CULL_BACK_FACING_TRIANGLES,
    //OPTIX_RAY_FLAG_NONE,
    for (float i = 0.f; i < 20.f; i+=0.05f ) {

    float3 origin = ray_origin + ray_direction*i;

    optixTrace(
        params.handle,
        origin,
        ray_direction,
        0.,
        1e16f,
        0.0f,
        OptixVisibilityMask( 1 ),
        OPTIX_RAY_FLAG_NONE,
        0, // SBT offset
        0, // SBT stride
        0, // missSBTIndex
        float3_as_args(payload_rgb));

    }
   

    params.frame_buffer[image_index] = make_color( payload_rgb );
    
}