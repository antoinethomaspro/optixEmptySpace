#include "hip/hip_runtime.h"
//
// Copyright (c) 2021, NVIDIA CORPORATION. All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions
// are met:
//  * Redistributions of source code must retain the above copyright
//    notice, this list of conditions and the following disclaimer.
//  * Redistributions in binary form must reproduce the above copyright
//    notice, this list of conditions and the following disclaimer in the
//    documentation and/or other materials provided with the distribution.
//  * Neither the name of NVIDIA CORPORATION nor the names of its
//    contributors may be used to endorse or promote products derived
//    from this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
// EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
// PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
// CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
// EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
// PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
// PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
// OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
// (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
//

#include <hip/hip_vector_types.h>
#include <optix_device.h>
#include "optixWhitted.h"
#include "random.h"
#include "helpers.h"
#include <cuda/helpers.h>


extern "C" {
__constant__ Params params;
}

extern "C" __global__ void __raygen__pinhole_camera()
{
    const uint3 idx = optixGetLaunchIndex();

    const CameraData* camera = (CameraData*) optixGetSbtDataPointer();

    const unsigned int image_index = params.width * idx.y + idx.x;

    float2 subpixel_jitter = make_float2(0.5f, 0.5f) ;

    float2 d = ((make_float2(idx.x, idx.y) + subpixel_jitter) / make_float2(params.width, params.height)) * 2.f - 1.f;
    float3 ray_origin = camera->eye;
    float3 ray_direction = normalize(d.x*camera->U + d.y*camera->V + camera->W);

    float distanceMin = -1.0f; // Front face intersection distance, stays negative when missed.
    float distanceMax = -1.0f; // Back  face intersection distance, stays negative when missed.

  
    unsigned int payload = __float_as_uint(distanceMin);
    optixTrace(
        params.handle2,                     // handle
        ray_origin,                         // float3 rayOrigin
        ray_direction,                      // float3 rayDirection
        0.f,                                 // float tmin
        1e16f,                              // float tmax
        0.0f,                               // float rayTime
        OptixVisibilityMask( 1 ),
        OPTIX_RAY_FLAG_CULL_BACK_FACING_TRIANGLES,
        1,                          // SBT offset (1 = CH2)
        RAY_TYPE_COUNT,             // SBT stride
        RAY_TYPE_RADIANCE,          // missSBTIndex 
        payload);


    distanceMin = __uint_as_float(payload);

    
    payload = __float_as_uint(distanceMax);
    optixTrace(
        params.handle2,                     // handle
        ray_origin,                         // float3 rayOrigin
        ray_direction,                      // float3 rayDirection
        0.f,                                 // float tmin
        1e16f,                              // float tmax
        0.0f,                               // float rayTime
        OptixVisibilityMask( 1 ),
        OPTIX_RAY_FLAG_CULL_FRONT_FACING_TRIANGLES,
        1,                          // SBT offset (1 = CH2)
        RAY_TYPE_COUNT,             // SBT stride
        RAY_TYPE_RADIANCE,          // missSBTIndex 
        payload);

    distanceMax = __uint_as_float(payload);



     if (0.0f < distanceMin && 0.0f < distanceMax)
        {
            if (distanceMin < distanceMax)
            {
            // The standard case: The ray started outside a volume and hit a front face and a back face farther away.
            // No special handling required. Use the two distance values as begin and end of the ray marching.
            }
            else // if distanceMin >= distanceMax
            {
            // This means a backface was hit before a father away front face.
            // In that case the ray origin must be inside a volume.
            
            distanceMin = 0.0f;
            }
        }
        else 
        {
            // Both rays missed, nothing to do here, fill per output buffer with default default data and return.
            params.frame_buffer[image_index] = make_color( make_float3(0.f) ); //why do I need this default outputbuffer???
            return;
        }

    //      else // if (0.0f < distanceMin && distanceMax < 0.0f)
    //         {
    //             // Illegal case. Front face hit but back face missed
    //             // This would mean there is no end to the volume and the ray marching would be until world end.
    //             // Maybe tint the result in some debug color to see if this happens.
    //             return;
    //         }

    float3 position;
    float3 payload_rgb = make_float3(0.f, 0.f, 0.f);
    // distanceMin = 0.f;
    // distanceMax = 10.f;
    // test transfert function min/max 
    // if alpha = 0, update distanceMin, distanceMax
    // if alpha > 0, pointQuery => look up for variance in selected range
    for (float distance = distanceMin; distance < distanceMax; distance += 0.1)
    {
        const float3 position = ray_origin + ray_direction * distance;

        optixTrace(
        params.handle,                     // handle
        position,                         // float3 rayOrigin
        ray_direction,                      // float3 rayDirection
        0.f,                                 // float tmin
        1e16f,                              // float tmax
        0.0f,                               // float rayTime
        OptixVisibilityMask( 1 ),
        OPTIX_RAY_FLAG_NONE,
        0,                          // SBT offset (1 = CH2)
        RAY_TYPE_COUNT,             // SBT stride
        RAY_TYPE_RADIANCE,          // missSBTIndex 
        float3_as_args(payload_rgb));
    }


    
    params.frame_buffer[image_index] = make_color( payload_rgb );
  

}





// extern "C" __global__ void __raygen__pinhole_camera()
// {
//     const uint3 idx = optixGetLaunchIndex();

//     const CameraData* camera = (CameraData*) optixGetSbtDataPointer();

//     const unsigned int image_index = params.width * idx.y + idx.x;

//     float2 subpixel_jitter = make_float2(0.5f, 0.5f) ;

//     float2 d = ((make_float2(idx.x, idx.y) + subpixel_jitter) / make_float2(params.width, params.height)) * 2.f - 1.f;
//     float3 ray_origin = camera->eye;
//     float3 ray_direction = normalize(d.x*camera->U + d.y*camera->V + camera->W);

//     float3 position;
//     float3 payload_rgb = make_float3(0.f, 0.f, 0.f);
  
//     for (float distance = 0.f; distance < 20.f; distance += 0.05)
//     {
//         const float3 position = ray_origin + ray_direction * distance;

//         optixTrace(
//         params.handle,                     // handle
//         position,                         // float3 rayOrigin
//         ray_direction,                      // float3 rayDirection
//         0.f,                                 // float tmin
//         1e16f,                              // float tmax
//         0.0f,                               // float rayTime
//         OptixVisibilityMask( 1 ),
//         OPTIX_RAY_FLAG_NONE,
//         0,                          // SBT offset (1 = CH2)
//         RAY_TYPE_COUNT,             // SBT stride
//         RAY_TYPE_RADIANCE,          // missSBTIndex 
//         float3_as_args(payload_rgb));
//     }


    
//     params.frame_buffer[image_index] = make_color( payload_rgb );
  

// }